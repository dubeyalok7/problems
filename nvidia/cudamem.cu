#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void myKernel(int *d_arr) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    d_arr[idx] = d_arr[idx]*2;
}

int main(void) {
    int *h_arr, *d_arr;
    h_arr = (int *)malloc(10*sizeof(int));
    for (int i=0; i<10; ++i)
        h_arr[i] = i; // Or other values

    // Sends data to device
    hipMalloc((void**) &d_arr, 10*sizeof(int));
    hipMemcpy(d_arr, h_arr, 10*sizeof(int), hipMemcpyHostToDevice);

    // Runs kernel on device
    myKernel<<< 2, 5 >>>(d_arr);

    // Retrieves data from device
    hipMemcpy(h_arr, d_arr, 10*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i<10; ++i)
        printf("Post kernel value in h_arr[%d] is: %d\n", i,h_arr[i]);

    hipFree(d_arr);
    free(h_arr);
    return 0;
}
