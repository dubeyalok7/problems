#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#define N 512

__global__ void add(int *a, int *b, int *c){
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main(void) {
  int *a, *b, *c;
  int *da, *db, &dc;
  int size = N *sizeof(int);

  //Alloc space for device copies of a,b, c
  hipMalloc((void **)&da, size);
  hipMalloc((void **)&db, size);
  hipMalloc((void **)&dc, size);

  //Alloc space for host copies of a, b, c and setup input values
  a = (int *)malloc(size); random_ints(a, N);
  b = (int *)malloc(size); random_ints(b, N);
  c = (int *)malloc(size);

  //Copy inputs to device
  hipMemcpy(da, a, size, hipMemcpyHostToDevice);
  hipMemcpy(db, b, size, hipMemcpyHostToDevice);

  //Launch add() kernel on GPU with N blocks
  add<<<N, 1>>>(da, db, dc);

  //Copy result back to host
  hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);

  free(a); free(b); free(c);
  hipFree(da); hipFree(db); hipFree(dc);
  return 0;
}
