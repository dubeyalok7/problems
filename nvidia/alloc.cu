#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c)
{
  *c = *a + *b;
}

int main(void)
{
  int a, b, c;
  int *d_a, *d_b, *d_c;
  int size = sizeof(int);

  //Allocate space for device copies of a, b, c
  hipMalloc((void**)&d_a, size);
  hipMalloc((void**)&d_b, size);
  hipMalloc((void**)&d_c, size);

  a = 2;
  b = 7;

  //copy inputs to device
  cudeMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  cudeMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  add<<<1,1>>>(d_a, d_b, d_c);

  //copy result back to host
  cudeMemcpy(&c, d_c, hipMemcpyDeviceToHost);

  //cleanup
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
