
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(void) {
  printf("Hello kernel\n");
}

int main(void) {
  mykernel<<<2,2>>>();
  printf("Hello World!\n");
  return 0;
}
